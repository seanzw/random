// nvcc -std=c++17 -arch=sm_120 -O3 bandwidth_test.cu
#include "benchmark_framework.cuh"
#include "cp_kernels.cuh"
#include "tma_kernels.cuh"

// Kernel wrapper for TMA
struct TMAKernelWrapper {
  template <int Stages, int CHUNK_BYTES, int REPEAT>
  void set_shmem_size(size_t shmem_bytes) {
    hipFuncSetAttribute(reinterpret_cast<const void*>(tma_bw_kernel<Stages), CHUNK_BYTES, REPEAT>,
                         hipFuncAttributeMaxDynamicSharedMemorySize,
                         shmem_bytes);
  }

  template <int Stages, int CHUNK_BYTES, int REPEAT>
  void launch(dim3 grid, dim3 block, size_t shmem_bytes, const uint8_t *src,
              unsigned long long *sink, size_t total_bytes) {
    tma_bw_kernel<Stages, CHUNK_BYTES, REPEAT>
        <<<grid, block, shmem_bytes>>>(src, sink, total_bytes);
  }
};

// Kernel wrapper for cp with configurable producer warps
template <int NUM_PRODUCER_WARPS = 1, bool USE_CP_ASYNC = true>
struct CPKernelWrapper {
  template <int Stages, int CHUNK_BYTES, int REPEAT>
  void set_shmem_size(size_t shmem_bytes) {
    hipFuncSetAttribute(reinterpret_cast<const void*>(cp_bw_kernel<Stages), CHUNK_BYTES, REPEAT,
                                      NUM_PRODUCER_WARPS, USE_CP_ASYNC>,
                         hipFuncAttributeMaxDynamicSharedMemorySize,
                         shmem_bytes);
  }

  template <int Stages, int CHUNK_BYTES, int REPEAT>
  void launch(dim3 grid, dim3 block, size_t shmem_bytes, const uint8_t *src,
              unsigned long long *sink, size_t total_bytes) {
    cp_bw_kernel<Stages, CHUNK_BYTES, REPEAT, NUM_PRODUCER_WARPS, USE_CP_ASYNC>
        <<<grid, block, shmem_bytes>>>(src, sink, total_bytes);
  }
};

// Kernel wrapper for normal load (using cp kernel with USE_CP_ASYNC=false)
template <int NUM_PRODUCER_WARPS = 1>
using NormalLoadKernelWrapper = CPKernelWrapper<NUM_PRODUCER_WARPS, false>;

int main() {
  const size_t total_bytes = size_t(64) * 1024 * 1024; // 64 MiB
  constexpr int repeat = 16;

  TestData test_data(total_bytes);
  const dim3 grid(test_data.get_num_sms());

  // TMA benchmarks (64 threads: 2 warps)
  const dim3 tma_block(64);
  printf("\n=== TMA Bandwidth Test ===\n");
  TMAKernelWrapper tma_wrapper;
  BandwidthBenchmark tma_bench(test_data, grid, tma_block, tma_wrapper,
                               "Hopper TMA bulk + forward/backward mbarrier");

  tma_bench.run_all_stages<256, repeat>();
  tma_bench.run_all_stages<512, repeat>();
  tma_bench.run_all_stages<1024, repeat>();
  tma_bench.run_all_stages<2048, repeat>();
  tma_bench.run_all_stages<4096, repeat>();
  tma_bench.run_all_stages<8192, repeat>();
  tma_bench.run_all_stages<16384, repeat>();

  // cp.async benchmarks with 1 producer warp (64 threads: 2 warps)
  const dim3 cp_async_1_block(64);
  printf("\n=== cp.async Bandwidth Test (1 Producer Warp) ===\n");
  CPKernelWrapper<1, true> cp_async_wrapper_1;
  BandwidthBenchmark cp_async_bench_1(test_data, grid, cp_async_1_block,
                                      cp_async_wrapper_1,
                                      "cp.async + 1 producer warp");

  cp_async_bench_1.run_all_stages<256, repeat>();
  cp_async_bench_1.run_all_stages<512, repeat>();
  cp_async_bench_1.run_all_stages<1024, repeat>();
  cp_async_bench_1.run_all_stages<2048, repeat>();
  cp_async_bench_1.run_all_stages<4096, repeat>();
  cp_async_bench_1.run_all_stages<8192, repeat>();
  cp_async_bench_1.run_all_stages<16384, repeat>();

  // cp.async benchmarks with 2 producer warps (96 threads: 3 warps)
  const dim3 cp_async_2_block(96);
  printf("\n=== cp.async Bandwidth Test (2 Producer Warps) ===\n");
  CPKernelWrapper<2, true> cp_async_wrapper_2;
  BandwidthBenchmark cp_async_bench_2(test_data, grid, cp_async_2_block,
                                      cp_async_wrapper_2,
                                      "cp.async + 2 producer warps");

  using power_of_two_sequence_2 = power_of_two_sequence<2, 32>;
  cp_async_bench_2.run_all_stages<256, repeat, power_of_two_sequence_2>();
  cp_async_bench_2.run_all_stages<512, repeat, power_of_two_sequence_2>();
  cp_async_bench_2.run_all_stages<1024, repeat, power_of_two_sequence_2>();
  cp_async_bench_2.run_all_stages<2048, repeat, power_of_two_sequence_2>();
  cp_async_bench_2.run_all_stages<4096, repeat, power_of_two_sequence_2>();
  cp_async_bench_2.run_all_stages<8192, repeat, power_of_two_sequence_2>();
  cp_async_bench_2.run_all_stages<16384, repeat, power_of_two_sequence_2>();

  // cp.async benchmarks with 4 producer warps (160 threads: 5 warps)
  const dim3 cp_async_4_block(160);
  using power_of_two_sequence_4 = power_of_two_sequence<4, 32>;
  printf("\n=== cp.async Bandwidth Test (4 Producer Warps) ===\n");
  CPKernelWrapper<4, true> cp_async_wrapper_4;
  BandwidthBenchmark cp_async_bench_4(test_data, grid, cp_async_4_block,
                                      cp_async_wrapper_4,
                                      "cp.async + 4 producer warps");

  cp_async_bench_4.run_all_stages<256, repeat, power_of_two_sequence_4>();
  cp_async_bench_4.run_all_stages<512, repeat, power_of_two_sequence_4>();
  cp_async_bench_4.run_all_stages<1024, repeat, power_of_two_sequence_4>();
  cp_async_bench_4.run_all_stages<2048, repeat, power_of_two_sequence_4>();
  cp_async_bench_4.run_all_stages<4096, repeat, power_of_two_sequence_4>();
  cp_async_bench_4.run_all_stages<8192, repeat, power_of_two_sequence_4>();
  cp_async_bench_4.run_all_stages<16384, repeat, power_of_two_sequence_4>();

  // Normal load benchmarks with 1 producer warp (64 threads: 2 warps)
  const dim3 normal_load_1_block(64);
  printf("\n=== Normal Load Bandwidth Test (1 Producer Warp) ===\n");
  NormalLoadKernelWrapper<1> normal_load_wrapper_1;
  BandwidthBenchmark normal_load_bench_1(test_data, grid, normal_load_1_block,
                                         normal_load_wrapper_1,
                                         "Normal load + 1 producer warp");

  normal_load_bench_1.run_all_stages<256, repeat>();
  normal_load_bench_1.run_all_stages<512, repeat>();
  normal_load_bench_1.run_all_stages<1024, repeat>();
  normal_load_bench_1.run_all_stages<2048, repeat>();
  normal_load_bench_1.run_all_stages<4096, repeat>();
  normal_load_bench_1.run_all_stages<8192, repeat>();
  normal_load_bench_1.run_all_stages<16384, repeat>();

  // Normal load benchmarks with 2 producer warps (96 threads: 3 warps)
  const dim3 normal_load_2_block(96);
  printf("\n=== Normal Load Bandwidth Test (2 Producer Warps) ===\n");
  NormalLoadKernelWrapper<2> normal_load_wrapper_2;
  BandwidthBenchmark normal_load_bench_2(test_data, grid, normal_load_2_block,
                                         normal_load_wrapper_2,
                                         "Normal load + 2 producer warps");

  normal_load_bench_2.run_all_stages<256, repeat, power_of_two_sequence_2>();
  normal_load_bench_2.run_all_stages<512, repeat, power_of_two_sequence_2>();
  normal_load_bench_2.run_all_stages<1024, repeat, power_of_two_sequence_2>();
  normal_load_bench_2.run_all_stages<2048, repeat, power_of_two_sequence_2>();
  normal_load_bench_2.run_all_stages<4096, repeat, power_of_two_sequence_2>();
  normal_load_bench_2.run_all_stages<8192, repeat, power_of_two_sequence_2>();
  normal_load_bench_2.run_all_stages<16384, repeat, power_of_two_sequence_2>();

  // Normal load benchmarks with 4 producer warps (160 threads: 5 warps)
  const dim3 normal_load_4_block(160);
  printf("\n=== Normal Load Bandwidth Test (4 Producer Warps) ===\n");
  NormalLoadKernelWrapper<4> normal_load_wrapper_4;
  BandwidthBenchmark normal_load_bench_4(test_data, grid, normal_load_4_block,
                                         normal_load_wrapper_4,
                                         "Normal load + 4 producer warps");

  normal_load_bench_4.run_all_stages<256, repeat, power_of_two_sequence_4>();
  normal_load_bench_4.run_all_stages<512, repeat, power_of_two_sequence_4>();
  normal_load_bench_4.run_all_stages<1024, repeat, power_of_two_sequence_4>();
  normal_load_bench_4.run_all_stages<2048, repeat, power_of_two_sequence_4>();
  normal_load_bench_4.run_all_stages<4096, repeat, power_of_two_sequence_4>();
  normal_load_bench_4.run_all_stages<8192, repeat, power_of_two_sequence_4>();
  normal_load_bench_4.run_all_stages<16384, repeat, power_of_two_sequence_4>();

  // Normal load benchmarks with 8 producer warps (288 threads: 9 warps)
  const dim3 normal_load_8_block(288);
  using power_of_two_sequence_8 = power_of_two_sequence<8, 32>;
  printf("\n=== Normal Load Bandwidth Test (8 Producer Warps) ===\n");
  NormalLoadKernelWrapper<8> normal_load_wrapper_8;
  BandwidthBenchmark normal_load_bench_8(test_data, grid, normal_load_8_block,
                                         normal_load_wrapper_8,
                                         "Normal load + 8 producer warps");

  normal_load_bench_8.run_all_stages<256, repeat, power_of_two_sequence_8>();
  normal_load_bench_8.run_all_stages<512, repeat, power_of_two_sequence_8>();
  normal_load_bench_8.run_all_stages<1024, repeat, power_of_two_sequence_8>();
  normal_load_bench_8.run_all_stages<2048, repeat, power_of_two_sequence_8>();
  normal_load_bench_8.run_all_stages<4096, repeat, power_of_two_sequence_8>();
  normal_load_bench_8.run_all_stages<8192, repeat, power_of_two_sequence_8>();
  normal_load_bench_8.run_all_stages<16384, repeat, power_of_two_sequence_8>();

  // Normal load benchmarks with 16 producer warps (544 threads: 17 warps)
  const dim3 normal_load_16_block(544);
  using power_of_two_sequence_16 = power_of_two_sequence<16, 32>;
  printf("\n=== Normal Load Bandwidth Test (16 Producer Warps) ===\n");
  NormalLoadKernelWrapper<16> normal_load_wrapper_16;
  BandwidthBenchmark normal_load_bench_16(test_data, grid, normal_load_16_block,
                                          normal_load_wrapper_16,
                                          "Normal load + 16 producer warps");

  normal_load_bench_16.run_all_stages<256, repeat, power_of_two_sequence_16>();
  normal_load_bench_16.run_all_stages<512, repeat, power_of_two_sequence_16>();
  normal_load_bench_16.run_all_stages<1024, repeat, power_of_two_sequence_16>();
  normal_load_bench_16.run_all_stages<2048, repeat, power_of_two_sequence_16>();
  normal_load_bench_16.run_all_stages<4096, repeat, power_of_two_sequence_16>();
  normal_load_bench_16.run_all_stages<8192, repeat, power_of_two_sequence_16>();
  normal_load_bench_16.run_all_stages<16384, repeat, power_of_two_sequence_16>();

  return 0;
}